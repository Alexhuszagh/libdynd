#include "hip/hip_runtime.h"
//
// Copyright (C) 2011-15 DyND Developers
// BSD 2-Clause License, see LICENSE.txt
//

#include <dynd/kernels/ckernel_builder.hpp>

__global__ void dynd::cuda_device_destroy(ckernel_prefix *self) { self->destroy(); }
